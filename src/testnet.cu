#include "hip/hip_runtime.h"
/**
 * Program for testing the Silk NNUE by running it in evaluation
 * mode on a set of test vectors
 */

#include <silk/nnue.hpp>
#include <stdio.h>
#include <iostream>


__global__ void testnet_kernel(const float4* nnue, const uint32_t* samples, float* output) {

    uint32_t signature = samples[blockIdx.x * 32 + threadIdx.x] & 255;
    float loss = kc::evaluate_nnue(signature, nnue);
    if (threadIdx.x == 0) { output[blockIdx.x] = loss; }

}


int main(int argc, char* argv[]) {

    if (argc != 5) {
        std::cerr << "Usage: ./testnet nnue.dat samples.dat output.dat n_samples" << std::endl;
        return 1;
    }

    int n_samples = std::stoll(argv[4]);

    float4* nnue_d;
    uint32_t* samples_d;
    float* outputs_d;

    float4* nnue_h;
    uint32_t* samples_h;
    float* outputs_h;

    hipMalloc((void**) &nnue_d, 3826176);
    hipMalloc((void**) &samples_d, 128 * n_samples);
    hipMalloc((void**) &outputs_d, 4 * n_samples);

    hipHostMalloc((void**) &nnue_h, 3826176);
    hipHostMalloc((void**) &samples_h, 128 * n_samples);
    hipHostMalloc((void**) &outputs_h, 4 * n_samples);

    {
        // load NNUE:
        FILE *fptr = fopen(argv[1], "r");
        fread(nnue_h, 512, 7473, fptr);
        fclose(fptr);
    }

    {
        // load samples:
        FILE *fptr = fopen(argv[2], "r");
        fread(samples_h, 128, n_samples, fptr);
        fclose(fptr);
    }

    hipMemcpy(nnue_d, nnue_h, 3826176, hipMemcpyHostToDevice);
    hipMemcpy(samples_d, samples_h, 128 * n_samples, hipMemcpyHostToDevice);
    testnet_kernel<<<n_samples, 32>>>(nnue_d, samples_d, outputs_d);
    hipMemcpy(outputs_h, outputs_d, 4 * n_samples, hipMemcpyDeviceToHost);

    {
        // save outputs:
        FILE* fptr = fopen(argv[3], "w");
        fwrite(outputs_h, 4, n_samples, fptr);
        fclose(fptr);
    }

    hipFree(nnue_d);
    hipFree(samples_d);
    hipFree(outputs_d);

    hipHostFree(nnue_d);
    hipHostFree(samples_d);
    hipHostFree(outputs_d);

    return 0;

}
