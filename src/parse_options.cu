#include "hip/hip_runtime.h"
#include "common.hpp"
#include "../cxxopts/include/cxxopts.hpp"

int main(int argc, char* argv[]) {

    if (argc < 1) {
        std::cerr << "Error: no command-line arguments were provided." << std::endl;
        return 1;
    }

    std::string silk_filename = argv[0];
    std::string nnue_filename = silk_filename + "_nnue.dat";

    // define argument parser
    cxxopts::Options options("silk", "A CUDA drifter searcher");

    options.add_options()

    // positional arguments (obligatory)
    ("input_filename", "LifeHistory RLE specifying the problem", cxxopts::value<std::string>())
    ("max_active_width", "maximum width of active region", cxxopts::value<int>())
    ("max_active_height", "maximum height of active region", cxxopts::value<int>())
    ("max_active_cells", "maximum number of active cells", cxxopts::value<int>())

    // optional arguments
    ("cadicals", "number of CaDiCaL threads to stabilise results", cxxopts::value<int>()->default_value("8"))
    ("p,period", "minimum period of oscillators to report", cxxopts::value<int>()->default_value("999999999"))
    ("d,dataset", "filename of dataset to output", cxxopts::value<std::string>()->default_value(""))
    ("m,max-perturbed-time", "maximum time the catalyst can interact", cxxopts::value<int>()->default_value("999999999"))            
    ("s,min-stable", "minimum unclean catalyst stable time before report", cxxopts::value<int>()->default_value("999999999"))
    ("e,exempt-existing", "whether to ignore constraints on ZOI of input catalyst cells", cxxopts::value<bool>()->default_value("false"))      

    // help
    ("h,help", "Print usage");

    options.parse_positional({"input_filename", "max_active_width", "max_active_height", "max_active_cells"});
    options.positional_help("path/to/input.rle max_active_width max_active_height max_active_cells");

    // apply argument parser to cmdline args
    auto result = options.parse(argc, argv);

    if (result.count("help")) {
        // print help message and exit
        std::cerr << options.help() << std::endl;
        return 0;
    }

    // extract positional arguments
    std::string input_filename = result["input_filename"].as<std::string>();
    int active_width = result["max_active_width"].as<int>();
    int active_height = result["max_active_height"].as<int>();
    int active_pop = result["max_active_cells"].as<int>();

    // extract optional arguments
    int num_cadical_threads = result["cadicals"].as<int>();
    int min_report_period = result["period"].as<int>();
    std::string dataset_filename = result["dataset"].as<std::string>();
    int max_perturbed_time = result["max-perturbed-time"].as<int>();
    int min_stable = result["min-stable"].as<int>();
    bool exempt_existing = result["exempt-existing"].as<bool>();

    std::cerr << "Info: Silk invoked as " << silk_filename << std::endl;

    // run program
    int return_code = silk_main(
        active_width,
        active_height,
        active_pop,
        input_filename,
        nnue_filename,
        num_cadical_threads,
        min_report_period,
        max_perturbed_time,
        min_stable,
        exempt_existing,
        dataset_filename
    );

    return return_code;
}

