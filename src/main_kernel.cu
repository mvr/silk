#include "hip/hip_runtime.h"
#include "common.hpp"
#include <silk/mainloop.hpp>
#include <silk/readrle.hpp>

/**
 * Main kernel that does the majority of the work.
 */
__global__ void __launch_bounds__(32, 16) computecellorbackup(

        // device-side pointers:
        const uint4* ctx, // common context for all problems
        uint4* prb, // problem ring buffer
        uint4* srb, // solution ring buffer
        int32_t* smd, // solution metadata
        uint64_t* global_counters,
        float4* nnue,

        // buffer sizes:
        uint32_t prb_size,
        uint32_t srb_size,

        // problem parameters:
        int max_width,
        int max_height,
        int max_pop,
        int rollout_gens,

        // miscellaneous:
        int min_period,
        uint32_t epsilon_threshold
    ) {

    // We use 5504 bytes (344 uint4s) to represent a pair of problems:
    //  -- bytes [0:5120]: stable information
    //      -- bytes [0:1024]: upper-right quadrant
    //      -- bytes [1024:2048]: lower-left quadrant
    //      -- bytes [2048:3072]: lower-right quadrant
    //      -- bytes [3072:4096]: upper-left quadrant in first problem
    //      -- bytes [4096:5120]: upper-left quadrant in second problem
    //  -- bytes [5120:5248]: metadata
    //  -- bytes [5248:5376]: active perturbation
    //  -- bytes [5376:5504]: signature of cell where split occurred
    constexpr uint64_t uint4s_per_pp = 344;

    // determine which problem to load:
    uint32_t logical_block_idx = 0;
    if (threadIdx.x == 0) {
        logical_block_idx = hh::atomic_add(global_counters + COUNTER_READING_HEAD, 1) & (prb_size - 1);
    }
    logical_block_idx = hh::shuffle_32(logical_block_idx, 0);
    uint4* reading_location = prb + uint4s_per_pp * (logical_block_idx >> 1);
    uint32_t block_parity = logical_block_idx & 1;

    // ********** LOAD PROBLEM **********

    // load problem metadata:
    uint32_t* problem_metadata_location = ((uint32_t*) (reading_location + 320));
    uint32_t metadata_y = problem_metadata_location[threadIdx.x];

    if (hh::shuffle_32(metadata_y, block_parity) == 0) {
        // we store problems in pairs for memory compression; however,
        // initially we only inject one problem so we want the ability
        // to have elements of the pair early-exit:
        return;
    }

    // load active perturbation:
    uint32_t perturbation = problem_metadata_location[threadIdx.x + 32];

    // load stable information:
    uint4 ad0, ad1, ad2, al2, al3, ad4, ad5, ad6;
    kc::load4(reading_location,       ad0.y, ad1.y, ad2.y, al2.y);
    kc::load4(reading_location + 32,  al3.y, ad4.y, ad5.y, ad6.y);
    kc::load4(reading_location + 64,  ad0.z, ad1.z, ad2.z, al2.z);
    kc::load4(reading_location + 96,  al3.z, ad4.z, ad5.z, ad6.z);
    kc::load4(reading_location + 128, ad0.w, ad1.w, ad2.w, al2.w);
    kc::load4(reading_location + 160, al3.w, ad4.w, ad5.w, ad6.w);
    kc::load4(reading_location + 192, ad0.x, ad1.x, ad2.x, al2.x);
    kc::load4(reading_location + 224, al3.x, ad4.x, ad5.x, ad6.x);

    // load stator constraints and shift into the correct reference frame:
    uint32_t px = hh::shuffle_32(metadata_y, 2);
    uint32_t py = hh::shuffle_32(metadata_y, 3);

    {
        uint32_t best_p = hh::shuffle_32(metadata_y, 30);
        uint32_t contrib = hh::shuffle_32(metadata_y, 31);
        if (block_parity & 1) { contrib ^= 255; }
        uint32_t this_cell = (threadIdx.x == (best_p >> 5)) ? (1u << (best_p & 31)) : 0u;
        if (contrib &   1) { ad0.x |= this_cell; }
        if (contrib &   2) { ad1.x |= this_cell; }
        if (contrib &   4) { ad2.x |= this_cell; }
        if (contrib &   8) { al2.x |= this_cell; }
        if (contrib &  16) { al3.x |= this_cell; }
        if (contrib &  32) { ad4.x |= this_cell; }
        if (contrib &  64) { ad5.x |= this_cell; }
        if (contrib & 128) { ad6.x |= this_cell; }
    }

    uint4 stator = ctx[threadIdx.x];
    kc::shift_torus_inplace(stator, -px, -py);

    // ********** INITIALISE SHARED MEMORY **********

    __shared__ uint32_t smem[256];
    __shared__ uint32_t metrics[32];
    #pragma unroll
    for (int i = 0; i < 8; i++) {
        smem[32*i + threadIdx.x] = 0;
    }
    metrics[threadIdx.x] = 0;
    __syncthreads();

    // ********** PERFORM PROPAGATION AND ADVANCING **********

    int return_code = kc::mainloop(
        ad0, ad1, ad2, al2, al3, ad4, ad5, ad6, stator,
        perturbation, px, py, max_width, max_height, max_pop, rollout_gens,
        smem, metrics
    );

    if (return_code == -1) { kc::bump_counter<true>(metrics, METRIC_DEADEND); }
    if (return_code ==  0) { kc::bump_counter<true>(metrics, METRIC_FIZZLE); }
    if (return_code ==  1) { kc::bump_counter<true>(metrics, METRIC_RESTAB); }
    if (return_code >=  2) { kc::bump_counter<true>(metrics, METRIC_OSCILLATOR); }

    // ********** HANDLE POTENTIAL SOLUTIONS **********

    if ((return_code >= 1) && (return_code < min_period)) {
        // we have found a restabilisation or oscillator of period
        // lower than the threshold, so this does not count as a
        // solution; modify the return code accordingly:
        return_code = -1;
    }

    __syncthreads();

    if (return_code >= -1) {
        // we have found a solution or a contradiction

        // flush metrics:
        hh::atomic_add(global_counters + threadIdx.x, metrics[threadIdx.x]);

        if (return_code >= 0) {
            // we have found a solution; write it out:
            uint32_t solution_idx = 0;
            if (threadIdx.x == 0) {
                solution_idx = hh::atomic_add(global_counters + COUNTER_SOLUTION_HEAD, 1) & (srb_size - 1);
                smd[solution_idx] = return_code;
            }
            solution_idx = hh::shuffle_32(solution_idx, 0);
            uint4* solution_location = srb + 256 * ((uint64_t) solution_idx);
            kc::shift_torus_inplace(ad0, px, py);
            solution_location[threadIdx.x] = ad0;
            kc::shift_torus_inplace(ad1, px, py);
            solution_location[threadIdx.x + 32] = ad1;
            kc::shift_torus_inplace(ad2, px, py);
            solution_location[threadIdx.x + 64] = ad2;
            kc::shift_torus_inplace(al2, px, py);
            solution_location[threadIdx.x + 96] = al2;
            kc::shift_torus_inplace(al3, px, py);
            solution_location[threadIdx.x + 128] = al3;
            kc::shift_torus_inplace(ad4, px, py);
            solution_location[threadIdx.x + 160] = ad4;
            kc::shift_torus_inplace(ad5, px, py);
            solution_location[threadIdx.x + 192] = ad5;
            kc::shift_torus_inplace(ad6, px, py);
            solution_location[threadIdx.x + 224] = ad6;
        }
        return;
    }

    // ********** WRITE BEGINNING OF OUTPUT PROBLEM PAIR **********

    uint32_t output_idx = 0;
    if (threadIdx.x == 0) {
        output_idx = hh::atomic_add(global_counters + COUNTER_WRITING_HEAD, 2) & (prb_size - 1);
    }
    output_idx = hh::shuffle_32(output_idx, 0);
    uint4* writing_location = prb + uint4s_per_pp * (output_idx >> 1);

    uint32_t total_info = 0; // between 0 and 32768
    total_info += kc::save4(writing_location,       ad0.y, ad1.y, ad2.y, al2.y);
    total_info += kc::save4(writing_location + 32,  al3.y, ad4.y, ad5.y, ad6.y);
    total_info += kc::save4(writing_location + 64,  ad0.z, ad1.z, ad2.z, al2.z);
    total_info += kc::save4(writing_location + 96,  al3.z, ad4.z, ad5.z, ad6.z);
    total_info += kc::save4(writing_location + 128, ad0.w, ad1.w, ad2.w, al2.w);
    total_info += kc::save4(writing_location + 160, al3.w, ad4.w, ad5.w, ad6.w);
    total_info += kc::save4(writing_location + 192, ad0.x, ad1.x, ad2.x, al2.x);
    total_info += kc::save4(writing_location + 224, al3.x, ad4.x, ad5.x, ad6.x);
    total_info = hh::warp_add(total_info);

    __syncthreads();

    // ********** PERFORM HARD BRANCHING DECISION **********

    uint32_t metadata_z = 0;

    uint32_t metadata_out = 0;

    float final_loss = kc::hard_branch(
        writing_location, perturbation, metadata_z, metadata_out,
        ad0.x, ad1.x, ad2.x, al2.x, al3.x, ad4.x, ad5.x, ad6.x, stator.x,
        max_width, max_height, max_pop, smem, epsilon_threshold,
        [&](uint32_t signature) __attribute__((always_inline)) {
            float loss = kc::evaluate_nnue(signature, nnue);
            return loss;
        }, metrics
    );

    // ********** WRITE OUTPUT PROBLEM METADATA **********

    uint32_t final_loss_bits = __float_as_int(final_loss);

    if (threadIdx.x < 2) { metadata_out = 1; }
    if (threadIdx.x == 2) { metadata_out = px; }
    if (threadIdx.x == 3) { metadata_out = py; }
    if (threadIdx.x == 4) { metadata_out = final_loss_bits; }
    if (threadIdx.x == 5) { metadata_out = total_info; }

    if (threadIdx.x < 8) {
        // copy the 32-byte metadata sector into the parent problem:
        problem_metadata_location[threadIdx.x + 8 + 8 * block_parity] = metadata_out;
    }

    uint32_t* solution_metadata_location = ((uint32_t*) (writing_location + 320));
    solution_metadata_location[threadIdx.x] = metadata_out;
    solution_metadata_location[threadIdx.x + 32] = perturbation;
    solution_metadata_location[threadIdx.x + 64] = metadata_z;

    // flush metrics:
    __syncthreads();
    hh::atomic_add(global_counters + threadIdx.x, metrics[threadIdx.x]);
}

/**
 * Rather trivial kernel that produces training data from the
 * output of computecellorbackup().
 */
__global__ void makennuedata(const uint4* prb, const uint64_t* global_counters, uint32_t* dataset, uint32_t prb_size) {

    constexpr uint64_t uint4s_per_pp = 344;

    // get the location from which to read:
    uint32_t pair_idx = global_counters[COUNTER_READING_HEAD] >> 1;
    pair_idx -= (blockIdx.x + 1);
    pair_idx &= ((prb_size >> 1) - 1);

    __shared__ uint32_t metadata[32];

    // load the metadata:
    const uint32_t* metadata_location = ((const uint32_t*) (prb + pair_idx * uint4s_per_pp + 320));
    metadata[threadIdx.x] = metadata_location[threadIdx.x];
    __syncthreads();

    uint32_t signature = metadata_location[threadIdx.x + 64];
    if (threadIdx.x == 29) { signature = 0; }
    if (threadIdx.x == 30) { signature = metadata[4]; }
    if (threadIdx.x == 31) {
        float total_loss = 0.0f;
        uint32_t info_0 = metadata[5];
        if (metadata[8]) {
            uint32_t info_gain = hh::min(metadata[13] - info_0, ((uint32_t) 20));
            float sub_loss = __int_as_float(metadata[12]);
            sub_loss = hh::max(0.0f, hh::min(sub_loss, 1.0f));
            total_loss += 0.375f + 0.125f * sub_loss - 0.015625f * info_gain;
        }
        if (metadata[16]) {
            uint32_t info_gain = hh::min(metadata[21] - info_0, ((uint32_t) 20));
            float sub_loss = __int_as_float(metadata[20]);
            sub_loss = hh::max(0.0f, hh::min(sub_loss, 1.0f));
            total_loss += 0.375f + 0.125f * sub_loss - 0.015625f * info_gain;
        }
        signature = __float_as_int(total_loss);
    }

    __syncthreads();
    dataset[pair_idx * 32 + threadIdx.x] = signature;
}

struct SilkGPU {

    // device-side pointers:
    uint4* ctx;
    uint4* prb; // problem ring buffer
    uint4* srb; // solution ring buffer
    int32_t* smd; // solution metadata
    uint64_t* global_counters;
    float4* nnue;
    uint32_t* dataset;

    // host-side pointers:
    uint64_t* host_counters;

    // buffer sizes:
    uint32_t prb_size;
    uint32_t srb_size;

    // problem parameters:
    int max_width;
    int max_height;
    int max_pop;
    int rollout_gens;

    SilkGPU(uint64_t prb_capacity, uint64_t srb_capacity) {
        hipMalloc((void**) &ctx, 512);
        hipMalloc((void**) &prb, 2752 * prb_capacity);
        hipMalloc((void**) &dataset, 64 * prb_capacity);
        hipMalloc((void**) &srb, 4096 * srb_capacity);
        hipMalloc((void**) &smd, 4 * srb_capacity);
        hipMalloc((void**) &global_counters, 512);
        hipMalloc((void**) &nnue, 7627264);

        hipHostMalloc((void**) &host_counters, 512);

        prb_size = prb_capacity;
        srb_size = srb_capacity;

        for (int i = 0; i < 64; i++) { host_counters[i] = 0; }
        host_counters[COUNTER_WRITING_HEAD] = 2;

        hipMemcpy(global_counters, host_counters, 512, hipMemcpyHostToDevice);

        hipMemset(ctx, 0, 512);
        hipMemset(nnue, 0, 7627264);

        max_width = 5;
        max_height = 5;
        max_pop = 10;
        rollout_gens = 6;
    }

    ~SilkGPU() {
        hipFree(ctx);
        hipFree(prb);
        hipFree(srb);
        hipFree(smd);
        hipFree(global_counters);
        hipFree(nnue);
        hipFree(dataset);
        hipHostFree(host_counters);
    }

    void inject_problem(std::vector<uint32_t> problem, std::vector<uint32_t> stator) {
        hipMemcpy(ctx, &(stator[0]), 512, hipMemcpyHostToDevice);
        hipMemcpy(prb, &(problem[0]), 5504, hipMemcpyHostToDevice);
    }

    void run_main_kernel(int blocks_to_launch, int min_period, double epsilon) {

        // we convert the probability epsilon into an integer in [0, 2**22]
        // as that is what the kernel expects:
        uint32_t epsilon_threshold = ((uint32_t) (epsilon * 4194304.0));

        // run the kernel:
        computecellorbackup<<<blocks_to_launch, 32>>>(
            ctx, prb, srb, smd, global_counters, nnue,
            prb_size, srb_size,
            max_width, max_height, max_pop, rollout_gens,
            min_period, epsilon_threshold
        );

        // extract training data into contiguous gmem:
        makennuedata<<<blocks_to_launch / 2, 32>>>(
            prb, global_counters, dataset, prb_size
        );

        hipMemcpy(host_counters, global_counters, 512, hipMemcpyDeviceToHost);
    }
};

void print_solution(const uint32_t* solution) {

    uint64_t tmp[512];
    for (int z = 0; z < 8; z++) {
        for (int y = 0; y < 32; y++) {
            tmp[64 * z + y]      = solution[128 * z + 4 * y    ] | (((uint64_t) solution[128 * z + 4 * y + 1]) << 32);
            tmp[64 * z + y + 32] = solution[128 * z + 4 * y + 2] | (((uint64_t) solution[128 * z + 4 * y + 3]) << 32);
        }
    }

    auto res = kc::complete_still_life(tmp, 4, true);

    if (res.size() == 0) { return; }

    for (int y = 0; y < 64; y++) {
        for (int x = 0; x < 64; x++) {
            std::cout << (((res[y] >> x) & 1) ? '*' : '.');
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main(int argc, char* argv[]) {

    kc::ProblemHolder ph("examples/tl.rle");
    auto problem = ph.swizzle_problem();
    auto stator = ph.swizzle_stator();

    SilkGPU silk(262144, 16384);

    silk.inject_problem(problem, stator);

    int problems = 2;

    for (int j = 0; j < 120; j++) {
        silk.run_main_kernel(problems, 11, 1.0);
        for (int i = 0; i < 64; i++) {
            std::cout << silk.host_counters[i] << " ";
        }
        std::cout << std::endl;
        problems = silk.host_counters[COUNTER_WRITING_HEAD] - silk.host_counters[COUNTER_READING_HEAD];
        if (problems == 0) { break; }
    }

    uint64_t solcount = silk.host_counters[COUNTER_SOLUTION_HEAD];

    return 0;

    if (solcount > 0) {
        uint32_t* host_solutions;
        int32_t* host_smd;
        hipHostMalloc((void**) &host_solutions, 4096 * solcount);
        hipHostMalloc((void**) &host_smd, 4 * solcount);

        hipMemcpy(host_solutions, silk.srb, 4096 * solcount, hipMemcpyDeviceToHost);
        hipMemcpy(host_smd, silk.smd, 4 * solcount, hipMemcpyDeviceToHost);

        for (uint64_t i = 0; i < solcount; i++) {
            std::cout << "***** found object with return code " << host_smd[i] << " *****" << std::endl;
            print_solution(host_solutions + 1024 * i);
        }

        hipFree(host_solutions);
        hipFree(host_smd);
    }

    return 0;
}
