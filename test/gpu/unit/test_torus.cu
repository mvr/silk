#include "hip/hip_runtime.h"
#include <silk/torus.hpp>
#include <cpads/random/prng.hpp>
#include <gtest/gtest.h>


__global__ void torus_shift_kernel(const uint4 *src, uint4 *dst) {

    int i = blockIdx.x & 63;
    int j = blockIdx.x >> 6;

    uint4 a = src[blockIdx.x * 32 + threadIdx.x];
    uint4 b = kc::shift_torus(a, i, j);

    dst[blockIdx.x * 32 + threadIdx.x] = b;

}


__global__ void plane_shift_kernel(const uint4 *src, uint4 *dst) {

    uint4 a = src[blockIdx.x * 32 + threadIdx.x];
    uint4 b;

    b.x = kc::shift_plane<false,  1>(a.x);
    b.y = kc::shift_plane<true,   1>(a.y);
    b.z = kc::shift_plane<false, -1>(a.z);
    b.w = kc::shift_plane<true,  -1>(a.w);

    dst[blockIdx.x * 32 + threadIdx.x] = b;
}


void check_shift(uint32_t* h_a, uint32_t* h_b, int x, int y) {

    uint64_t a[64];
    uint64_t b[64];

    for (int i = 0; i < 32; i++) {
        a[i]      = h_a[4*i  ] + (((uint64_t) h_a[4*i+1]) << 32);
        a[i + 32] = h_a[4*i+2] + (((uint64_t) h_a[4*i+3]) << 32);
        b[i]      = h_b[4*i  ] + (((uint64_t) h_b[4*i+1]) << 32);
        b[i + 32] = h_b[4*i+2] + (((uint64_t) h_b[4*i+3]) << 32);
    }

    for (int i = 0; i < 64; i++) {
        uint64_t c = a[i];
        if (x != 0) { c = (c << x) | (c >> (64-x)); }
        EXPECT_EQ(c, b[(i + y) & 63]);
    }
}


TEST(Plane, GetMiddle) {

    EXPECT_EQ(kc::get_middle(0x00ffff00u), ((uint32_t) 0));
    EXPECT_EQ(kc::get_middle(0x0ffff000u), ((uint32_t) 4));

}


TEST(Plane, Shift) {

    constexpr int n = 4096;
    uint4* d_a;
    uint4* d_b;
    uint32_t* h_a;
    uint32_t* h_b;

    hipMalloc((void**) &d_a, n);
    hipMalloc((void**) &d_b, n);
    hipHostMalloc((void**) &h_a, n);
    hipHostMalloc((void**) &h_b, n);

    hh::PRNG pcg(1, 2, 3);

    for (int i = 0; i < (n >> 2); i++) {
        // generate some random data
        h_a[i] = pcg.generate();
    }

    hipMemcpy(d_a, h_a, n, hipMemcpyHostToDevice);
    plane_shift_kernel<<<(n / 512), 32>>>(d_a, d_b);
    hipMemcpy(h_b, d_b, n, hipMemcpyDeviceToHost);

    for (int i = 0; i < (n >> 2); i += 128) {
        for (int j = 0; j < 32; j++) {
            EXPECT_EQ(h_b[i + 4*j + 0], h_a[i + 4*j + 0] << 1);
            EXPECT_EQ(h_b[i + 4*j + 2], h_a[i + 4*j + 2] >> 1);
            uint32_t c = 0;
            if (j >= 1) { c = h_a[i + 4*j + 1 - 4]; }
            EXPECT_EQ(h_b[i + 4*j + 1], c);
            uint32_t d = 0;
            if (j < 31) { d = h_a[i + 4*j + 3 + 4]; }
            EXPECT_EQ(h_b[i + 4*j + 3], d);
        }
    }

    hipFree(d_a);
    hipFree(d_b);
    hipHostFree(h_a);
    hipHostFree(h_b);
}


TEST(Torus, Shift) {

    constexpr int n = 512 * 64 * 64;
    uint4* d_a;
    uint4* d_b;
    uint32_t* h_a;
    uint32_t* h_b;

    hipMalloc((void**) &d_a, n);
    hipMalloc((void**) &d_b, n);
    hipHostMalloc((void**) &h_a, n);
    hipHostMalloc((void**) &h_b, n);

    hh::PRNG pcg(1, 2, 3);

    for (int i = 0; i < (n >> 2); i++) {
        // generate some random data
        h_a[i] = pcg.generate();
    }

    hipMemcpy(d_a, h_a, n, hipMemcpyHostToDevice);
    torus_shift_kernel<<<4096, 32>>>(d_a, d_b);
    hipMemcpy(h_b, d_b, n, hipMemcpyDeviceToHost);

    for (int j = 0; j < 63; j++) {
        for (int i = 0; i < 63; i++) {
            int k = j * 64 + i;
            check_shift(h_a + 128 * k, h_b + 128 * k, i, j);
        }
    }

    hipFree(d_a);
    hipFree(d_b);
    hipHostFree(h_a);
    hipHostFree(h_b);
}
